#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>

#include <algorithm>
#include <iostream>
#include <vector>

#include "HugeCTR/include/common.hpp"
#include "HugeCTR/include/data_simulator.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/data.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/frequent_embedding.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/model.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/utils.hpp"
#include "HugeCTR/include/tensor2.hpp"
#include "HugeCTR/include/utils.hpp"

namespace HugeCTR {

namespace hybrid_embedding {

template <typename dtype, typename emtype>
void FrequentEmbedding<dtype, emtype>::initialize_embedding_vectors() {
  CudaDeviceContext context(gpu_resource_->get_device_id());

  const size_t num_tables = data_.global_table_sizes.size();
  for (size_t i = 0; i < num_tables; i++) {
    float up_bound = sqrt(1.f / data_.global_table_sizes[i]);
    UniformGenerator::fill(
        frequent_embedding_vectors_[i], -up_bound, up_bound, gpu_resource_->get_sm_count(),
        gpu_resource_->get_replica_uniform_curand_generator(), gpu_resource_->get_stream());
  }
}
template <typename dtype, typename emtype>
__global__ void forward_network_frequent_embedding(
    uint32_t embedding_vec_size, uint32_t global_sample_index_base,
    const uint32_t* frequent_sample_indices,  // local
    const dtype* samples,                     // global
    const dtype* category_frequent_index,     // global
    const float* frequent_embedding_vectors,  // global
    emtype* interaction_layer_input)          // local
{
  int bid = blockIdx.x;   // each block corresponding to one category
  int tid = threadIdx.x;  // each thread corresponding to one element in the embedding vector
  uint32_t index = frequent_sample_indices[bid];
  dtype category = samples[index + global_sample_index_base];
  dtype frequent_index = category_frequent_index[category];
  interaction_layer_input[index * embedding_vec_size + tid] =
      frequent_embedding_vectors[frequent_index * embedding_vec_size + tid];
}

template <typename dtype, typename emtype>
void FrequentEmbedding<dtype, emtype>::forward_network(emtype* interaction_layer_input,
                                                       hipStream_t stream) {
  // concatenate the embedding vectors into the buffer for
  // top-mlp input

  // Kefeng: type here, use FrequentEmbedding::frequent_sample_indices
  // in short this is what it should do:
  //   for index in frequent_sample_indices:
  //      output[index][0..em_vec_size-1] =
  //      frequent_embedding_vectors_[category_frequent_index[samples[index]]][0..em_vec_size-1]

  // carefully check
  uint32_t samples_per_instance = data_.samples.get_num_elements() / model_.num_instances;
  uint32_t global_sample_index_base = model_.global_instance_id * samples_per_instance;

  const size_t block_size = embedding_vec_size_;
  const size_t grid_size = frequent_sample_indices_.get_num_elements();
  forward_network_frequent_embedding<<<grid_size, block_size, 0, stream>>>(
      embedding_vec_size_, global_sample_index_base, frequent_sample_indices_.get_ptr(),
      data_.samples.get_ptr(), model_.category_frequent_index.get_ptr(),
      frequent_embedding_vectors_block_.get_ptr(), interaction_layer_input);
}

template <typename dtype, typename emtype>
void FrequentEmbedding<dtype, emtype>::update_network() {
  // TODO: create update()
}

template <typename dtype, typename emtype>
void FrequentEmbedding<dtype, emtype>::update_model() {
  // TODO: create update()
}

template class FrequentEmbedding<uint32_t, __half>;
template class FrequentEmbedding<uint32_t, float>;
template class FrequentEmbedding<unsigned long, __half>;
template class FrequentEmbedding<unsigned long, float>;
}  // namespace hybrid_embedding

}  // namespace HugeCTR