#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>

#include <algorithm>
#include <iostream>
#include <vector>

#include "HugeCTR/include/common.hpp"
#include "HugeCTR/include/data_simulator.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/data.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/frequent_embedding.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/model.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/utils.hpp"
#include "HugeCTR/include/tensor2.hpp"
#include "HugeCTR/include/utils.hpp"

namespace HugeCTR {

namespace hybrid_embedding {

template <typename dtype, typename emtype>
void FrequentEmbedding<dtype, emtype>::initialize_embedding_vectors() {
  CudaDeviceContext context(gpu_resource_->get_device_id());

  const size_t num_tables = data_.global_table_sizes.size();
  for (size_t i = 0; i < num_tables; i++) {
    float up_bound = sqrt(1.f / data_.global_table_sizes[i]);
    UniformGenerator::fill(
        frequent_embedding_vectors_[i], -up_bound, up_bound, gpu_resource_->get_sm_count(),
        gpu_resource_->get_replica_uniform_curand_generator(), gpu_resource_->get_stream());
  }
}

template <typename dtype, typename emtype>
void FrequentEmbedding<dtype, emtype>::forward_network(const emtype *interaction_layer_input) {
  // concatenate the embedding vectors into the buffer for
  // top-mlp input

  // Kefeng: type here, use FrequentEmbedding::frequent_sample_indices
  // in short this is what it should do:
  //   for index in frequent_sample_indices:
  //      output[index][0..em_vec_size-1] =
  //      frequent_embedding_vectors_[category_frequent_index[samples[index]]][0..em_vec_size-1]
}

template <typename dtype, typename emtype>
void FrequentEmbedding<dtype, emtype>::update_network() {
  // TODO: create update()
}

template <typename dtype, typename emtype>
void FrequentEmbedding<dtype, emtype>::update_model() {
  // TODO: create update()
}

template class FrequentEmbedding<uint32_t, __half>;
template class FrequentEmbedding<uint32_t, float>;
template class FrequentEmbedding<unsigned long, __half>;
template class FrequentEmbedding<unsigned long, float>;
}  // namespace hybrid_embedding

}  // namespace HugeCTR