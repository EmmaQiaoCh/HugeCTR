/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "hybrid_embedding_utils.hpp"

#include <vector>
#include <iostream>


template <typename dtype>
void download_tensor(std::vector<dtype>& h_tensor, Tensor2<dtype> tensor, CudaStream_t stream) {
  CK_CUDA_THROW(hipStreamSynchronize(stream));
  CK_CUDA_THROW(hipMemcpy(
    h_tensor.data(), tensor.get_ptr(), 
    tensor.get_size_in_bytes(), hipMemcpyDeviceToHost, stream)); 
  CK_CUDA_THROW(hipStreamSynchronize(stream));
}


template <typename dtype>
void upload_tensor(std::vector<dtype>& h_tensor, Tensor2<dtype> tensor, CudaStream_t stream) {
  CK_CUDA_THROW(hipStreamSynchronize(stream));
  CK_CUDA_THROW(hipMemcpyAsync(
    tensor.get_ptr(), h_tensor.data(), 
    h_tensor.size()*sizeof(dtype), hipMemcpyHostToDevice, stream));
  CK_CUDA_THROW(hipStreamSynchronize(stream));
}


/// @brief flatten_samples converts the member variable 'data' and store 
///        the result in member variable 'samples'. 
///        Per network, the columns corresponding to embedding tables 
///        are concatenated and categories get an unique index / label.
template <typename dtype>
void HybridEmbeddingData::flatten_samples(

    hipStream_t stream
) {

  std::cout << "WARNING: flatten_samples needs to be placed on the GPU!" << std::endl;
  // TODO : perform conversion by kernel (before start of iteration ? => see below)
  //        for batch_size = 55*1024
  //        batch_size * 26 * 4 / 1600e9 = 3.67 microseconds, 
  // 
  // Remark:
  //        Doesn't need to be before start of kernel. 
  //        Would be nice to have just before calculating indices, since
  //        those would be in L2 cache already.
  std::vector<dtype> h_data;
  download_tensor<dtype>(h_data, data, stream);

  const size_t num_tables = table_sizes.size();
  std::vector<dtype> embedding_offsets(num_tables);
  dtype embedding_offset = (dtype) 0;
  for (size_t embedding = 0; embedding < num_tables; ++embedding) {
    embedding_offsets[embedding] = embedding_offset;
    embedding_offset += table_sizes[embedding];
  }

  // keep order of input samples, convert each data field such that categories
  // from different categorical features have different label / index
  size_t indx = 0;
  std::vector<dtype> h_samples(num_tables * batch_size);
  for (size_t i = 0; i < network_batch_size; ++i) {
    for (size_t embedding=0; embedding < num_tables; ++embedding) {
      h_samples[indx] = h_data[indx] + embedding_offsets[embedding];
      indx++;
    }
  }

  // TODO : remove
  upload_tensor(h_samples, samples, stream);
}


/// @brief init_model calculates the optimal number of frequent categories 
///        given the calibration of the all-to-all and all-reduce.
template<dtype>
void HybridEmbeddingModel::init_model(
    const CalibrationData& calibration,
    const HybridEmbeddingData<dtype>& embedding_data
) {

  if (calibration.all_to_all_times.size() > 0) {
    // calibration is given, perform fully optimized hybrid model
    CK_THROW(Error_t::WrongInput, "initialization hybrid model from communication calibration not available yet");
  } else {
      Tensor2<dtype> samples = embedding_data.samples;
      size_t num_nodes = (double) num_gpus_per_node.size();

      // Use threshold to determine number of frequent categories,
      // calculates optimal number of frequent categories when the all-to-all 
      // and all-reduce are both bandwidth limited.
      double all_reduce_bandwidth = calibration.max_all_reduce_bandwidth;
      double all_to_all_bandwidth = calibration.max_all_to_all_bandwidth;
      n_threshold = all_to_all_bandwidth / all_reduce_bandwidth 
                    * (double) num_nodes / ((double) num_nodes - 1.);

      // samples, num_samples, categories_sorted, counts_sorted

      // sort samples by category
      // per category get count
      // sort category by count

      sort_categories_by_count(
          samples, num_samples, categories_sorted, counts_sorted);

      

  }
}

// template definitions
#include "HugeCTR/include/embeddings/hybrid_embedding/hybrid_embedding_utils_include.cuh"
