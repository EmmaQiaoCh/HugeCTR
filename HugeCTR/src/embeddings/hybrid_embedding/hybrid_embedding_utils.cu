#include "hybrid_embedding_utils.hpp"
#include "HugeCTR/include/embedding/hybrid_sparse_embedding.hpp"

#include <vector>


template <typename dtype>
void download_tensor(std::vector<dtype>& h_tensor, Tensor2<dtype> tensor, CudaStream_t stream) {
  CK_CUDA_THROW(hipStreamSynchronize(stream));
  CK_CUDA_THROW(hipMemcpy(
    h_tensor.data(), tensor.get_ptr(), 
    tensor.get_size_in_bytes(), hipMemcpyDeviceToHost, stream)); 
  CK_CUDA_THROW(hipStreamSynchronize(stream));
}


template <typename dtype>
void upload_tensor(std::vector<dtype>& h_tensor, Tensor2<dtype> tensor, CudaStream_t stream) {
  CK_CUDA_THROW(hipStreamSynchronize(stream));
  CK_CUDA_THROW(hipMemcpyAsync(
    tensor.get_ptr(), h_tensor.data(), 
    h_tensor.size()*sizeof(dtype), hipMemcpyHostToDevice, stream));
  CK_CUDA_THROW(hipStreamSynchronize(stream));
}


/// @brief flatten_samples converts the member variable 'data' and store 
///        the result in member variable 'samples'. 
///        Per network, the columns corresponding to embedding tables 
///        are concatenated and categories get an unique index / label.
template <typename dtype>
void HybridEmbeddingData::flatten_samples(hipStream_t stream) {

  std::cout << "WARNING: flatten_samples needs to be placed on the GPU!" << std::endl;
  // TODO : perform conversion by kernel (before start of iteration ? => see below)
  //        for batch_size = 55*1024
  //        batch_size * 26 * 4 / 1600e9 = 3.67 microseconds, 
  // 
  // Remark:
  //        Doesn't need to be before start of kernel. 
  //        Would be nice to have just before calculating indices, since
  //        those would be in L2 cache already.
  std::vector<dtype> h_data;
  download_tensor<dtype>(h_data, data, stream);

  const size_t num_tables = table_sizes.size();
  std::vector<dtype> embedding_offsets(num_tables);
  dtype embedding_offset = (dtype) 0;
  for (size_t embedding = 0; embedding < num_tables; ++embedding) {
    embedding_offsets[embedding] = embedding_offset;
    embedding_offset += table_sizes[embedding];
  }

  uint32_t network_batch_size = batch_size / num_networks;

  std::vector<dtype> h_samples(num_tables * batch_size);
  for (size_t network=0; network < num_networks; ++network) {
    size_t data_offset = network * network_batch_size * num_tables;
    for (size_t i = 0; i < network_batch_size; ++i) {
      for (size_t embedding=0; embedding < num_tables; ++embedding) {
        dtype category_offset = embedding_offsets[embedding];
        h_samples[indx] = h_data[data_offset + i*num_tables + embedding] + category_offset;
        indx++;
      }
    }
  }

  upload_tensor(h_samples, samples, stream);
}


/// @brief init_model calculates the optimal number of frequent categories 
///        given the calibration of the all-to-all and all-reduce.
template<dtype>
void HybridEmbeddingModel::init_model(
    const CalibrationInitializationData& calibration,
    const HybridEmbeddingData<dtype>& embedding_data
) {

  if (calibration.all_to_all_times.size() > 0) {
    // calibration is given, perform fully optimized hybrid model
    CK_THROW(Error_t::WrongInput, "initialization hybrid model from communication calibration not available yet");
  } else {
      Tensor2<dtype> samples = embedding_data.samples;
      size_t num_nodes = (double) num_gpus_per_node.size();

      // Use threshold to determine number of frequent categories,
      // calculates optimal number of frequent categories when the all-to-all 
      // and all-reduce are both bandwidth limited.
      double all_reduce_bandwidth = calibration.max_all_reduce_bandwidth;
      double all_to_all_bandwidth = calibration.max_all_to_all_bandwidth;
      n_threshold = all_to_all_bandwidth / all_reduce_bandwidth 
                    * (double) num_nodes / ((double) num_nodes - 1.);

      

      sort_categories_by_count(samples, num_samples, categories_sorted, counts_sorted);
  }
}

// template definitions
#include "HugeCTR/include/embeddings/hybrid_embedding/hybrid_embedding_utils_include.cuh"
