#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <iostream>
#include <utility>
#include <vector>

#include "HugeCTR/include/embeddings/hybrid_embedding/infrequent_embedding.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/model.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/data.hpp"
#include "HugeCTR/include/embeddings/hybrid_embedding/utils.hpp"
#include "HugeCTR/include/utils.cuh"


namespace hybrid_embedding {


template <typename dtype, typename emtype>
void InfrequentEmbedding::initialize_embedding_vectors() {
  // TODO: create initialize_embedding_vectors()
}


template <typename T>
static bool lesser_by_first(const std::pair<T, T>& a, const std::pair<T, T>& b) {
  return (a.first < b.first);
}


/// TODO: GPU version
template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::calculate_model_indices(hipStream_t stream) {
  std::cout << "WARNING: calculate_model_indices must be done on GPU!" << std::endl;

  size_t local_batch_size = ceildiv<size_t>(data_.batch_size, data_.num_networks);
  const size_t num_tables = data_.table_sizes.size();

  std::vector<dtype> h_samples;
  download_tensor<dtype>(h_samples, data_.samples, stream);
  std::vector<dtype> h_category_location;
  download_tensor<dtype>(h_category_location, model_.category_location, stream);

  std::vector<uint32_t> h_model_indices = std::vector<dtype>(data_.batch_size * num_tables);
  std::vector<uint32_t> h_model_indices_offsets = std::vector<dtype>(data_.num_networks + 1);

  // Prefix sum
  size_t sum = 0;
  for (size_t j = 0; j < data_.batch_size; j++) {
    if (j % local_batch_size == 0) {
      h_model_indices_offsets[j / local_batch_size] = sum;
    }
    for (size_t i = 0; i < num_tables; i++) {
      size_t idx = j * num_tables + i;

      dtype category = h_samples[idx];
      dtype network_id = h_category_location[2 * category];
      bool mask = network_id == model_.global_network_id;

      sum += static_cast<size_t>(mask);

      if (mask) h_model_indices[sum - 1] = idx;
    }
  }
  // Total size stored at the end of the offsets vector
  h_model_indices_offsets[data_.num_networks] = sum;

  upload_tensor(h_model_indices, model_indices_, stream);
  upload_tensor(h_model_indices_offsets, model_indices_offsets_, stream);
}


/// TODO: GPU version
template <typename dtype, typename emtype>
void InfrequentEmbedding<dtype, emtype>::calculate_network_indices(hipStream_t stream) {
  std::cout << "WARNING: calculate_network_indices must be done on GPU!" << std::endl;

  size_t local_batch_size = ceildiv<size_t>(data_.batch_size, data_.num_networks);
  const size_t num_tables = data_.table_sizes.size();

  std::vector<dtype> h_samples;
  download_tensor<dtype>(h_samples, data_.samples, stream);
  std::vector<dtype> h_category_location;
  download_tensor<dtype>(h_category_location, model_.category_location, stream);

  std::vector<std::pair<uint32_t, uint32_t>> h_network_sources_indices =
      std::vector<std::pair<uint32_t, uint32_t>>(local_batch_size * num_tables);

  // Prefix sum only of this GPU's sub-batch
  size_t sum = 0;
  for (size_t j = local_batch_size * model_.global_network_id;
       j < std::min(data_.batch_size, local_batch_size * (model_.global_network_id + 1)); j++) {
    for (size_t i = 0; i < num_tables; i++) {
      size_t idx = j * num_tables + i;

      dtype category = h_samples[idx];
      dtype network_id = h_category_location[2 * category];
      bool mask = network_id < data_.num_networks;

      sum += static_cast<size_t>(mask);

      uint32_t local_mlp_index = (j - local_batch_size * model_.global_network_id) * num_tables + i;

      if (mask)
        h_network_sources_indices[sum - 1] =
            std::make_pair(static_cast<uint32_t>(model_.global_network_id), local_mlp_index);
    }
  }

  // Sort by source only, otherwise stable
  std::sort(h_network_sources_indices.begin(), h_network_sources_indices.begin() + sum,
            lesser_by_first<dtype>);

  // Retrieve indices
  std::vector<dtype> h_network_indices = std::vector<dtype>(local_batch_size * num_tables);
  for (size_t idx = 0; idx < sum; idx++) {
    h_network_indices[idx] = h_network_sources_indices[idx].second;
  }
  // Compute offsets
  std::vector<dtype> h_network_indices_offsets = std::vector<dtype>(data_.num_networks + 1);
  for (size_t i = 0; i < data_.num_networks; i++) {
    h_network_indices_offsets[i] =
        std::lower_bound(h_network_sources_indices.begin(), h_network_sources_indices.begin() + sum,
                         std::make_pair<uint32_t, uint32_t>(i, 0), lesser_by_first<uint32_t>) -
        h_network_sources_indices.begin();
  }
  // Total size stored at the end of the offsets vector
  h_network_indices_offsets[data_.num_networks] = sum;

  upload_tensor(h_network_indices, network_indices_, stream);
  upload_tensor(h_network_indices_offsets, network_indices_offsets_, stream);
}


template class InfrequentEmbedding<uint32_t, __half>;
template class InfrequentEmbedding<uint32_t, float>;
template class InfrequentEmbedding<size_t, __half>;
template class InfrequentEmbedding<size_t, float>;
}  // namespace HugeCTR
